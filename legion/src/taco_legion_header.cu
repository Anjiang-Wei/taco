#include "hip/hip_runtime.h"
#include "taco_legion_header.h"
#include "pitches.h"
#include "hipcub/hipcub.hpp"

using namespace Legion;

const int THREADS_PER_BLOCK = 256;

Domain RectCompressedPosPartitionDownwards::gputask(const Task *task, const std::vector<Legion::PhysicalRegion> &regions,
                                                    Legion::Context ctx, Runtime *runtime) {
  FieldID field = *(FieldID*)(task->args);
  Accessor acc(regions[0], field);
  auto dom = runtime->get_index_space_domain(ctx, regions[0].get_logical_region().get_index_space());
  taco_iassert(dom.dense());
  if (dom.empty()) {
    return Rect<1>::make_empty();
  }
  Rect<1> lo, hi;
  hipMemcpy(&lo, acc.ptr(dom.lo()), sizeof(Rect<1>), hipMemcpyHostToDevice);
  hipMemcpy(&hi, acc.ptr(dom.hi()), sizeof(Rect<1>), hipMemcpyHostToDevice);
  return Rect<1>{lo.lo, hi.hi};
}

template<int DIM, Legion::PrivilegeMode MODE>
using RCFYPAccessor = Legion::FieldAccessor<MODE, Legion::Rect<1>, DIM, Legion::coord_t, Realm::AffineAccessor<Legion::Rect<1>, DIM, Legion::coord_t>>;
template<int DIM>
__global__
void rectCompressedFinalizeYieldPositionsKernel(const Rect <DIM> fullBounds, const Rect <DIM> iterBounds,
                                                const Pitches<DIM - 1> pitches, size_t volume,
                                                RCFYPAccessor <DIM, READ_WRITE> output,
                                                RCFYPAccessor <DIM, READ_ONLY> ghost) {
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  auto point = pitches.unflatten(idx, iterBounds.lo);
  if (point == Point<DIM>::ZEROES()) {
    output[point].lo = 0;
  } else {
    output[point].lo = ghost[getPreviousPoint(point, fullBounds)].lo;
  }
}

void RectCompressedFinalizeYieldPositions::gputask(const Legion::Task *task,
                                                   const std::vector<Legion::PhysicalRegion> &regions, Legion::Context ctx,
                                                   Legion::Runtime *runtime) {
  auto output = regions[0];
  auto outputlr = output.get_logical_region();
  auto ghost = regions[1];
  std::vector<FieldID> fields;
  output.get_fields(fields);
  taco_iassert(runtime->has_parent_logical_partition(ctx, outputlr));
  auto outputPart = runtime->get_parent_logical_partition(ctx, outputlr);
  auto outputParent = runtime->get_parent_logical_region(ctx, outputPart);
  taco_iassert(fields.size() == 1);
  switch (outputlr.get_dim()) {
#define BLOCK(DIM) \
    case DIM: {    \
      Rect<DIM> fullBounds = runtime->get_index_space_domain(ctx, outputParent.get_index_space()).bounds<DIM, coord_t>(); \
      Rect<DIM> iterBounds = runtime->get_index_space_domain(ctx, outputlr.get_index_space()).bounds<DIM, coord_t>();     \
      Pitches<DIM - 1> pitches;                                                                                           \
      auto volume = pitches.flatten(iterBounds);                                                                          \
      auto blocks = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;                                                 \
      if (blocks != 0) {                                                                                                  \
        Accessor<DIM, READ_WRITE> outAcc(output, fields[0]); \
        Accessor<DIM, READ_ONLY> ghostAcc(ghost, fields[0]); \
        rectCompressedFinalizeYieldPositionsKernel<DIM><<<blocks, THREADS_PER_BLOCK>>>(fullBounds, iterBounds, pitches, volume, outAcc, ghostAcc); \
      }            \
      break;       \
    }
    LEGION_FOREACH_N(BLOCK)
#undef BLOCK
    default:
      taco_iassert(false);
  }
}

template<typename T, int DIM, Legion::PrivilegeMode MODE>
using RCGSEIAccessor = Legion::FieldAccessor<MODE, T, DIM, Legion::coord_t, Realm::AffineAccessor<T, DIM, Legion::coord_t>>;

template<int DIM>
__global__
void rectCompressedGetSeqInsertEdgesConstructFinalKernel(const Rect<DIM> iterBounds,
                                                         const Pitches<DIM - 1> pitches,
                                                         const size_t volume,
                                                         RCGSEIAccessor<Rect<1>, DIM, WRITE_ONLY> output,
                                                         DeferredBuffer<int64_t, DIM> scanBuf) {
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  auto point = pitches.unflatten(idx, iterBounds.lo);
  auto lo = (idx == 0) ? 0 : scanBuf[pitches.unflatten(idx - 1, iterBounds.lo)];
  auto hi = scanBuf[point] - 1;
  output[point] = {lo, hi};
}

template<int DIM>
int64_t RectCompressedGetSeqInsertEdges::scanBodyGPU(Context ctx, Runtime *runtime, Rect<DIM> iterBounds,
                                                     Accessor<Rect<1>, DIM, WRITE_ONLY> output,
                                                     Accessor<int64_t, DIM, READ_ONLY> input,
                                                     Memory::Kind tmpMemKind) {
  Pitches<DIM - 1> pitches;
  auto volume = pitches.flatten(iterBounds);
  if (volume == 0) {
    return 0;
  }

  int64_t initVal = 0;
  DeferredBuffer<int64_t, DIM> scanBuf(iterBounds, tmpMemKind, &initVal);

  auto inputPtrBot = input.ptr(iterBounds.lo);
  auto bufPtrBot = scanBuf.ptr(iterBounds.lo);

  // Perform the scan.
  void* tmpStorage = NULL;
  size_t tmpStorageBytes = 0;
  hipcub::DeviceScan::InclusiveSum(tmpStorage, tmpStorageBytes, inputPtrBot, bufPtrBot, volume);
  DeferredBuffer<char, 1> cubTmpStorage(Rect<1>(0, tmpStorageBytes - 1), tmpMemKind);
  hipcub::DeviceScan::InclusiveSum(cubTmpStorage.ptr(0), tmpStorageBytes, inputPtrBot, bufPtrBot, volume);

  // Construct the final result.
  auto blocks = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  if (blocks != 0) {
    rectCompressedGetSeqInsertEdgesConstructFinalKernel<DIM><<<blocks, THREADS_PER_BLOCK>>>(iterBounds, pitches, volume, output, scanBuf);
  }

  // Return the result of the scan.
  int64_t scanVal;
  hipMemcpy(&scanVal, scanBuf.ptr(iterBounds.hi), sizeof(int64_t), hipMemcpyHostToDevice);
  return scanVal;
}

int64_t RectCompressedGetSeqInsertEdges::scanTaskGPU(const Legion::Task *task,
                                                     const std::vector<Legion::PhysicalRegion> &regions,
                                                     Legion::Context ctx,
                                                     Legion::Runtime *runtime) {
  // Unpack arguments for the task.
  FieldID outputField, inputField;
  std::tie(outputField, inputField) = RectCompressedGetSeqInsertEdges::unpackScanTaskArgs(task);

  // Figure out what kind of memory body should allocate its temporary within.
  Memory::Kind tmpMemKind = Realm::Memory::GPU_FB_MEM;

  auto output = regions[0];
  auto input = regions[1];
  auto outputlr = output.get_logical_region();
  switch (outputlr.get_dim()) {
#define BLOCK(DIM) \
    case DIM: {    \
      Rect<DIM> iterBounds = runtime->get_index_space_domain(ctx, outputlr.get_index_space()).bounds<DIM, coord_t>();     \
      Accessor<Rect<1>, DIM, WRITE_ONLY> outAcc(output, outputField); \
      Accessor<int64_t, DIM, READ_ONLY> inAcc(input, inputField); \
      return RectCompressedGetSeqInsertEdges::scanBodyGPU<DIM>(ctx, runtime, iterBounds, outAcc, inAcc, tmpMemKind); \
    }
    LEGION_FOREACH_N(BLOCK)
#undef BLOCK
    default:
      taco_iassert(false);
      return 0; // Keep the compiler happy.
  }
}

template<int DIM>
__global__
void rectCompressedGetSeqInsertEdgesApplyPartialResultsKernel(
    const Rect<DIM> iterBounds,
    const Pitches<DIM - 1> pitches,
    const size_t volume,
    RCGSEIAccessor<Rect<1>, DIM, READ_WRITE> output,
    int64_t value
) {
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  output[pitches.unflatten(idx, iterBounds.lo)] += Point<1>(value);
}

void RectCompressedGetSeqInsertEdges::applyPartialResultsTaskGPU(const Legion::Task *task,
                                                                 const std::vector<Legion::PhysicalRegion> &regions,
                                                                 Legion::Context ctx, Legion::Runtime *runtime) {
  FieldID outputField;
  int64_t value;
  std::tie(outputField, value) = RectCompressedGetSeqInsertEdges::unpackApplyPartialResultsTaskArgs(task);

  auto output = regions[0];
  auto outputlr = output.get_logical_region();
  switch (outputlr.get_dim()) {
#define BLOCK(DIM) \
    case DIM: {    \
      Rect<DIM> iterBounds = runtime->get_index_space_domain(ctx, outputlr.get_index_space()).bounds<DIM, coord_t>();     \
      Accessor<Rect<1>, DIM, READ_WRITE> outAcc(output, outputField);                                                     \
      Pitches<DIM - 1> pitches;                                                                                           \
      auto volume = pitches.flatten(iterBounds);                                                                          \
      auto blocks = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;                                                 \
      if (blocks != 0) {                                                                                                  \
        rectCompressedGetSeqInsertEdgesApplyPartialResultsKernel<DIM><<<blocks, THREADS_PER_BLOCK>>>(iterBounds, pitches, volume, outAcc, value);             \
      }             \
      break;             \
    }
    LEGION_FOREACH_N(BLOCK)
#undef BLOCK
    default:
      taco_iassert(false);
  }
}
