#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "cudalibs.h"
#include "leaf_kernels.cuh"
#include "taco_legion_header.h"
#include "taco_mapper.h"
#define TACO_MIN(_a,_b) ((_a) < (_b) ? (_a) : (_b))
using namespace Legion;
typedef FieldAccessor<READ_ONLY,double,2,coord_t,Realm::AffineAccessor<double,2,coord_t>> AccessorROdouble2;
typedef ReductionAccessor<SumReduction<double>,true,2,coord_t,Realm::AffineAccessor<double,2,coord_t>> AccessorReducedouble2;

struct task_1Args {
  int32_t sfID;
  int32_t gridDim;
};
struct task_2Args {
  int32_t sfID;
  int32_t gridDim;
};
struct task_3Args {
  int32_t sfID;
  int32_t gridDim;
};
struct task_4Args {
  int32_t gridDim;
};

LogicalPartition partitionLegion(Context ctx, Runtime* runtime, LogicalRegion a, int32_t gridDim) {
  int a1_dimension = runtime->get_index_space_domain(get_index_space(a)).hi()[0] + 1;
  int a2_dimension = runtime->get_index_space_domain(get_index_space(a)).hi()[1] + 1;
  auto a_index_space = get_index_space(a);

  Point<2> lowerBound = Point<2>(0, 0);
  Point<2> upperBound = Point<2>((gridDim - 1), (gridDim - 1));
  auto distFusedIndexSpace = runtime->create_index_space(ctx, Rect<2>(lowerBound, upperBound));
  DomainT<2> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<2>(distFusedIndexSpace));
  auto aDomain = runtime->get_index_space_domain(ctx, a_index_space);
  DomainPointColoring aColoring = DomainPointColoring();
  for (PointInDomainIterator<2> itr = PointInDomainIterator<2>(domain); itr.valid(); itr++) {
    int32_t in = (*itr)[0];
    int32_t jn = (*itr)[1];
    Point<2> aStart = Point<2>((in * ((a1_dimension + (gridDim - 1)) / gridDim) + 0 / gridDim), (jn * ((a2_dimension + (gridDim - 1)) / gridDim) + 0 / gridDim));
    Point<2> aEnd = Point<2>(TACO_MIN((in * ((a1_dimension + (gridDim - 1)) / gridDim) + ((a1_dimension + (gridDim - 1)) / gridDim - 1)),aDomain.hi()[0]), TACO_MIN((jn * ((a2_dimension + (gridDim - 1)) / gridDim) + ((a2_dimension + (gridDim - 1)) / gridDim - 1)),aDomain.hi()[1]));
    Rect<2> aRect = Rect<2>(aStart, aEnd);
    if (!aDomain.contains(aRect.lo) || !aDomain.contains(aRect.hi)) {
      aRect = aRect.make_empty();
    }
    aColoring[(*itr)] = aRect;
  }
  auto aPartition = runtime->create_index_partition(ctx, a_index_space, domain, aColoring, LEGION_DISJOINT_COMPLETE_KIND);
  return runtime->get_logical_partition(ctx, get_logical_region(a), aPartition);
}

std::vector<LogicalPartition> partitionForplaceLegionA(Context ctx, Runtime* runtime, LogicalRegion a, int32_t gridDim) {
  int a1_dimension = runtime->get_index_space_domain(get_index_space(a)).hi()[0] + 1;
  int a2_dimension = runtime->get_index_space_domain(get_index_space(a)).hi()[1] + 1;
  auto a_index_space = get_index_space(a);

  Point<3> lowerBound = Point<3>(0, 0, 0);
  Point<3> upperBound = Point<3>((gridDim - 1), (gridDim - 1), 0);
  auto distFusedIndexSpace = runtime->create_index_space(ctx, Rect<3>(lowerBound, upperBound));
  DomainT<3> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<3>(distFusedIndexSpace));
  auto aDomain = runtime->get_index_space_domain(ctx, a_index_space);
  DomainPointColoring aColoring = DomainPointColoring();
  for (PointInDomainIterator<3> itr = PointInDomainIterator<3>(domain); itr.valid(); itr++) {
    int32_t in = (*itr)[0];
    int32_t jn = (*itr)[1];
    Point<2> aStart = Point<2>((in * ((a1_dimension + (gridDim - 1)) / gridDim) + 0 / gridDim), (jn * ((a2_dimension + (gridDim - 1)) / gridDim) + 0 / gridDim));
    Point<2> aEnd = Point<2>(TACO_MIN((in * ((a1_dimension + (gridDim - 1)) / gridDim) + ((a1_dimension + (gridDim - 1)) / gridDim - 1)),aDomain.hi()[0]), TACO_MIN((jn * ((a2_dimension + (gridDim - 1)) / gridDim) + ((a2_dimension + (gridDim - 1)) / gridDim - 1)),aDomain.hi()[1]));
    Rect<2> aRect = Rect<2>(aStart, aEnd);
    if (!aDomain.contains(aRect.lo) || !aDomain.contains(aRect.hi)) {
      aRect = aRect.make_empty();
    }
    aColoring[(*itr)] = aRect;
  }
  auto aPartition = runtime->create_index_partition(ctx, a_index_space, domain, aColoring, LEGION_COMPUTE_KIND);
  std::vector<LogicalPartition> computePartitions = std::vector<LogicalPartition>();
  computePartitions.push_back(runtime->get_logical_partition(ctx, get_logical_region(a), aPartition));
  return computePartitions;
}

void task_1(const Task* task, const std::vector<PhysicalRegion>& regions, Context ctx, Runtime* runtime) {
  PhysicalRegion a = regions[0];

  int32_t distFused = task->index_point[0];
  task_1Args* args = (task_1Args*)(task->args);
  int32_t gridDim = args->gridDim;


  int32_t in = getIndexPoint(task, 0);
  int32_t jn = getIndexPoint(task, 1);
  int32_t kn = getIndexPoint(task, 2);
}

void placeLegionA(Context ctx, Runtime* runtime, LogicalRegion a, LogicalPartition aPartition, int32_t gridDim) {

  Point<3> lowerBound = Point<3>(0, 0, 0);
  Point<3> upperBound = Point<3>((gridDim - 1), (gridDim - 1), 0);
  auto distFusedIndexSpace = runtime->create_index_space(ctx, Rect<3>(lowerBound, upperBound));
  DomainT<3> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<3>(distFusedIndexSpace));
  RegionRequirement aReq = RegionRequirement(aPartition, 0, READ_ONLY, EXCLUSIVE, get_logical_region(a));
  aReq.add_field(FID_VAL);
  std::vector<int> dims = std::vector<int>();
  dims.push_back(gridDim);
  dims.push_back(gridDim);
  dims.push_back(gridDim);
  registerPlacementShardingFunctor(ctx, runtime, shardingID(7), dims);
  task_1Args taskArgsRaw;
  taskArgsRaw.sfID = shardingID(7);
  taskArgsRaw.gridDim = gridDim;
  TaskArgument taskArgs = TaskArgument(&taskArgsRaw, sizeof(task_1Args));
  IndexLauncher launcher = IndexLauncher(taskID(1), domain, taskArgs, ArgumentMap());
  launcher.add_region_requirement(aReq);
  launcher.tag = TACOMapper::PLACEMENT_SHARD;
  auto fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();

}

std::vector<LogicalPartition> partitionForplaceLegionB(Context ctx, Runtime* runtime, LogicalRegion b, int32_t gridDim) {
  int b1_dimension = runtime->get_index_space_domain(get_index_space(b)).hi()[0] + 1;
  int b2_dimension = runtime->get_index_space_domain(get_index_space(b)).hi()[1] + 1;
  auto b_index_space = get_index_space(b);

  Point<3> lowerBound = Point<3>(0, 0, 0);
  Point<3> upperBound = Point<3>((gridDim - 1), 0, (gridDim - 1));
  auto distFusedIndexSpace = runtime->create_index_space(ctx, Rect<3>(lowerBound, upperBound));
  DomainT<3> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<3>(distFusedIndexSpace));
  auto bDomain = runtime->get_index_space_domain(ctx, b_index_space);
  DomainPointColoring bColoring = DomainPointColoring();
  for (PointInDomainIterator<3> itr = PointInDomainIterator<3>(domain); itr.valid(); itr++) {
    int32_t in = (*itr)[0];
    int32_t jn = (*itr)[2];
    Point<2> bStart = Point<2>((in * ((b1_dimension + (gridDim - 1)) / gridDim) + 0 / gridDim), (jn * ((b2_dimension + (gridDim - 1)) / gridDim) + 0 / gridDim));
    Point<2> bEnd = Point<2>(TACO_MIN((in * ((b1_dimension + (gridDim - 1)) / gridDim) + ((b1_dimension + (gridDim - 1)) / gridDim - 1)),bDomain.hi()[0]), TACO_MIN((jn * ((b2_dimension + (gridDim - 1)) / gridDim) + ((b2_dimension + (gridDim - 1)) / gridDim - 1)),bDomain.hi()[1]));
    Rect<2> bRect = Rect<2>(bStart, bEnd);
    if (!bDomain.contains(bRect.lo) || !bDomain.contains(bRect.hi)) {
      bRect = bRect.make_empty();
    }
    bColoring[(*itr)] = bRect;
  }
  auto bPartition = runtime->create_index_partition(ctx, b_index_space, domain, bColoring, LEGION_COMPUTE_KIND);
  std::vector<LogicalPartition> computePartitions = std::vector<LogicalPartition>();
  computePartitions.push_back(runtime->get_logical_partition(ctx, get_logical_region(b), bPartition));
  return computePartitions;
}

void task_2(const Task* task, const std::vector<PhysicalRegion>& regions, Context ctx, Runtime* runtime) {
  PhysicalRegion b = regions[0];

  int32_t distFused = task->index_point[0];
  task_2Args* args = (task_2Args*)(task->args);
  int32_t gridDim = args->gridDim;


  int32_t in = getIndexPoint(task, 0);
  int32_t kn = getIndexPoint(task, 1);
  int32_t jn = getIndexPoint(task, 2);
}

void placeLegionB(Context ctx, Runtime* runtime, LogicalRegion b, LogicalPartition bPartition, int32_t gridDim) {

  Point<3> lowerBound = Point<3>(0, 0, 0);
  Point<3> upperBound = Point<3>((gridDim - 1), 0, (gridDim - 1));
  auto distFusedIndexSpace = runtime->create_index_space(ctx, Rect<3>(lowerBound, upperBound));
  DomainT<3> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<3>(distFusedIndexSpace));
  RegionRequirement bReq = RegionRequirement(bPartition, 0, READ_ONLY, EXCLUSIVE, get_logical_region(b));
  bReq.add_field(FID_VAL);
  std::vector<int> dims = std::vector<int>();
  dims.push_back(gridDim);
  dims.push_back(gridDim);
  dims.push_back(gridDim);
  registerPlacementShardingFunctor(ctx, runtime, shardingID(9), dims);
  task_2Args taskArgsRaw;
  taskArgsRaw.sfID = shardingID(9);
  taskArgsRaw.gridDim = gridDim;
  TaskArgument taskArgs = TaskArgument(&taskArgsRaw, sizeof(task_2Args));
  IndexLauncher launcher = IndexLauncher(taskID(2), domain, taskArgs, ArgumentMap());
  launcher.add_region_requirement(bReq);
  launcher.tag = TACOMapper::PLACEMENT_SHARD;
  auto fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();

}

std::vector<LogicalPartition> partitionForplaceLegionC(Context ctx, Runtime* runtime, LogicalRegion c, int32_t gridDim) {
  int c1_dimension = runtime->get_index_space_domain(get_index_space(c)).hi()[0] + 1;
  int c2_dimension = runtime->get_index_space_domain(get_index_space(c)).hi()[1] + 1;
  auto c_index_space = get_index_space(c);

  Point<3> lowerBound = Point<3>(0, 0, 0);
  Point<3> upperBound = Point<3>(0, (gridDim - 1), (gridDim - 1));
  auto distFusedIndexSpace = runtime->create_index_space(ctx, Rect<3>(lowerBound, upperBound));
  DomainT<3> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<3>(distFusedIndexSpace));
  auto cDomain = runtime->get_index_space_domain(ctx, c_index_space);
  DomainPointColoring cColoring = DomainPointColoring();
  for (PointInDomainIterator<3> itr = PointInDomainIterator<3>(domain); itr.valid(); itr++) {
    int32_t in = (*itr)[1];
    int32_t jn = (*itr)[2];
    Point<2> cStart = Point<2>((in * ((c1_dimension + (gridDim - 1)) / gridDim) + 0 / gridDim), (jn * ((c2_dimension + (gridDim - 1)) / gridDim) + 0 / gridDim));
    Point<2> cEnd = Point<2>(TACO_MIN((in * ((c1_dimension + (gridDim - 1)) / gridDim) + ((c1_dimension + (gridDim - 1)) / gridDim - 1)),cDomain.hi()[0]), TACO_MIN((jn * ((c2_dimension + (gridDim - 1)) / gridDim) + ((c2_dimension + (gridDim - 1)) / gridDim - 1)),cDomain.hi()[1]));
    Rect<2> cRect = Rect<2>(cStart, cEnd);
    if (!cDomain.contains(cRect.lo) || !cDomain.contains(cRect.hi)) {
      cRect = cRect.make_empty();
    }
    cColoring[(*itr)] = cRect;
  }
  auto cPartition = runtime->create_index_partition(ctx, c_index_space, domain, cColoring, LEGION_COMPUTE_KIND);
  std::vector<LogicalPartition> computePartitions = std::vector<LogicalPartition>();
  computePartitions.push_back(runtime->get_logical_partition(ctx, get_logical_region(c), cPartition));
  return computePartitions;
}

void task_3(const Task* task, const std::vector<PhysicalRegion>& regions, Context ctx, Runtime* runtime) {
  PhysicalRegion c = regions[0];

  int32_t distFused = task->index_point[0];
  task_3Args* args = (task_3Args*)(task->args);
  int32_t gridDim = args->gridDim;


  int32_t kn = getIndexPoint(task, 0);
  int32_t in = getIndexPoint(task, 1);
  int32_t jn = getIndexPoint(task, 2);
}

void placeLegionC(Context ctx, Runtime* runtime, LogicalRegion c, LogicalPartition cPartition, int32_t gridDim) {

  Point<3> lowerBound = Point<3>(0, 0, 0);
  Point<3> upperBound = Point<3>(0, (gridDim - 1), (gridDim - 1));
  auto distFusedIndexSpace = runtime->create_index_space(ctx, Rect<3>(lowerBound, upperBound));
  DomainT<3> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<3>(distFusedIndexSpace));
  RegionRequirement cReq = RegionRequirement(cPartition, 0, READ_ONLY, EXCLUSIVE, get_logical_region(c));
  cReq.add_field(FID_VAL);
  std::vector<int> dims = std::vector<int>();
  dims.push_back(gridDim);
  dims.push_back(gridDim);
  dims.push_back(gridDim);
  registerPlacementShardingFunctor(ctx, runtime, shardingID(11), dims);
  task_3Args taskArgsRaw;
  taskArgsRaw.sfID = shardingID(11);
  taskArgsRaw.gridDim = gridDim;
  TaskArgument taskArgs = TaskArgument(&taskArgsRaw, sizeof(task_3Args));
  IndexLauncher launcher = IndexLauncher(taskID(3), domain, taskArgs, ArgumentMap());
  launcher.add_region_requirement(cReq);
  launcher.tag = TACOMapper::PLACEMENT_SHARD;
  auto fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();

}

std::vector<LogicalPartition> partitionForcomputeLegion(Context ctx, Runtime* runtime, LogicalRegion a, LogicalRegion b, LogicalRegion c, int32_t gridDim) {
  auto a_index_space = get_index_space(a);
  int b1_dimension = runtime->get_index_space_domain(get_index_space(b)).hi()[0] + 1;
  int b2_dimension = runtime->get_index_space_domain(get_index_space(b)).hi()[1] + 1;
  auto b_index_space = get_index_space(b);
  int c2_dimension = runtime->get_index_space_domain(get_index_space(c)).hi()[1] + 1;
  auto c_index_space = get_index_space(c);

  Point<3> lowerBound = Point<3>(0, 0, 0);
  Point<3> upperBound = Point<3>((gridDim - 1), (gridDim - 1), (gridDim - 1));
  auto distFusedIndexSpace = runtime->create_index_space(ctx, Rect<3>(lowerBound, upperBound));
  DomainT<3> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<3>(distFusedIndexSpace));
  auto aDomain = runtime->get_index_space_domain(ctx, a_index_space);
  auto bDomain = runtime->get_index_space_domain(ctx, b_index_space);
  auto cDomain = runtime->get_index_space_domain(ctx, c_index_space);
  DomainPointColoring aColoring = DomainPointColoring();
  DomainPointColoring bColoring = DomainPointColoring();
  DomainPointColoring cColoring = DomainPointColoring();
  for (PointInDomainIterator<3> itr = PointInDomainIterator<3>(domain); itr.valid(); itr++) {
    int32_t in = (*itr)[0];
    int32_t jn = (*itr)[1];
    int32_t kn = (*itr)[2];
    Point<2> aStart = Point<2>((in * ((b1_dimension + (gridDim - 1)) / gridDim) + 0 / gridDim), (jn * ((c2_dimension + (gridDim - 1)) / gridDim) + 0 / gridDim));
    Point<2> aEnd = Point<2>(TACO_MIN((in * ((b1_dimension + (gridDim - 1)) / gridDim) + ((b1_dimension + (gridDim - 1)) / gridDim - 1)),aDomain.hi()[0]), TACO_MIN((jn * ((c2_dimension + (gridDim - 1)) / gridDim) + ((c2_dimension + (gridDim - 1)) / gridDim - 1)),aDomain.hi()[1]));
    Rect<2> aRect = Rect<2>(aStart, aEnd);
    if (!aDomain.contains(aRect.lo) || !aDomain.contains(aRect.hi)) {
      aRect = aRect.make_empty();
    }
    aColoring[(*itr)] = aRect;
    Point<2> bStart = Point<2>((in * ((b1_dimension + (gridDim - 1)) / gridDim) + 0 / gridDim), (kn * ((b2_dimension + (gridDim - 1)) / gridDim) + 0 / gridDim));
    Point<2> bEnd = Point<2>(TACO_MIN((in * ((b1_dimension + (gridDim - 1)) / gridDim) + ((b1_dimension + (gridDim - 1)) / gridDim - 1)),bDomain.hi()[0]), TACO_MIN((kn * ((b2_dimension + (gridDim - 1)) / gridDim) + ((b2_dimension + (gridDim - 1)) / gridDim - 1)),bDomain.hi()[1]));
    Rect<2> bRect = Rect<2>(bStart, bEnd);
    if (!bDomain.contains(bRect.lo) || !bDomain.contains(bRect.hi)) {
      bRect = bRect.make_empty();
    }
    bColoring[(*itr)] = bRect;
    Point<2> cStart = Point<2>((kn * ((b2_dimension + (gridDim - 1)) / gridDim) + 0 / gridDim), (jn * ((c2_dimension + (gridDim - 1)) / gridDim) + 0 / gridDim));
    Point<2> cEnd = Point<2>(TACO_MIN((kn * ((b2_dimension + (gridDim - 1)) / gridDim) + ((b2_dimension + (gridDim - 1)) / gridDim - 1)),cDomain.hi()[0]), TACO_MIN((jn * ((c2_dimension + (gridDim - 1)) / gridDim) + ((c2_dimension + (gridDim - 1)) / gridDim - 1)),cDomain.hi()[1]));
    Rect<2> cRect = Rect<2>(cStart, cEnd);
    if (!cDomain.contains(cRect.lo) || !cDomain.contains(cRect.hi)) {
      cRect = cRect.make_empty();
    }
    cColoring[(*itr)] = cRect;
  }
  auto aPartition = runtime->create_index_partition(ctx, a_index_space, domain, aColoring, LEGION_ALIASED_COMPLETE_KIND);
  auto bPartition = runtime->create_index_partition(ctx, b_index_space, domain, bColoring, LEGION_ALIASED_COMPLETE_KIND);
  auto cPartition = runtime->create_index_partition(ctx, c_index_space, domain, cColoring, LEGION_ALIASED_COMPLETE_KIND);
  std::vector<LogicalPartition> computePartitions = std::vector<LogicalPartition>();
  computePartitions.push_back(runtime->get_logical_partition(ctx, get_logical_region(a), aPartition));
  computePartitions.push_back(runtime->get_logical_partition(ctx, get_logical_region(b), bPartition));
  computePartitions.push_back(runtime->get_logical_partition(ctx, get_logical_region(c), cPartition));
  return computePartitions;
}

void task_4(const Task* task, const std::vector<PhysicalRegion>& regions, Context ctx, Runtime* runtime) {
  PhysicalRegion a = regions[0];
  PhysicalRegion b = regions[1];
  PhysicalRegion c = regions[2];

  int32_t distFused = task->index_point[0];
  task_4Args* args = (task_4Args*)(task->args);
  int32_t gridDim = args->gridDim;

  auto a_index_space = get_index_space(a);
  auto b_index_space = get_index_space(b);
  auto c_index_space = get_index_space(c);
  AccessorROdouble2 b_vals(b, FID_VAL);
  AccessorROdouble2 c_vals(c, FID_VAL);
  AccessorReducedouble2 a_vals(a, FID_VAL, LEGION_REDOP_SUM_FLOAT64);

  int32_t in = getIndexPoint(task, 0);
  int32_t jn = getIndexPoint(task, 1);
  int32_t kn = getIndexPoint(task, 2);
  auto aDomain = runtime->get_index_space_domain(ctx, a_index_space);
  auto bDomain = runtime->get_index_space_domain(ctx, b_index_space);
  auto cDomain = runtime->get_index_space_domain(ctx, c_index_space);
  if (bDomain.get_volume() == 0 || cDomain.get_volume() == 0)
    return ;

  double alpha = 1.0000000000000000;
  hipblasHandle_t handle = getCuBLAS();
  hipStream_t taskStream = hipStream_t();
  hipStreamCreate(&(taskStream));
  CHECK_CUBLAS(hipblasSetStream(handle, taskStream));
  CHECK_CUBLAS(hipblasDgemm(
    handle,
    HIPBLAS_OP_N,
    HIPBLAS_OP_N,
    (1 + (cDomain.hi()[1] - cDomain.lo()[1])),
    (1 + (bDomain.hi()[0] - bDomain.lo()[0])),
    (1 + (cDomain.hi()[0] - cDomain.lo()[0])),
    &(alpha),
    c_vals.ptr(cDomain.lo()),
    (c_vals.accessor.strides[0] / sizeof(double)),
    b_vals.ptr(bDomain.lo()),
    (b_vals.accessor.strides[0] / sizeof(double)),
    &(alpha),
    a_vals.ptr(aDomain.lo()),
    (a_vals.accessor.strides[0] / sizeof(double))
  ));
}

void computeLegion(Context ctx, Runtime* runtime, LogicalRegion a, LogicalRegion b, LogicalRegion c, LogicalPartition aPartition, LogicalPartition bPartition, LogicalPartition cPartition, int32_t gridDim) {

  Point<3> lowerBound = Point<3>(0, 0, 0);
  Point<3> upperBound = Point<3>((gridDim - 1), (gridDim - 1), (gridDim - 1));
  auto distFusedIndexSpace = runtime->create_index_space(ctx, Rect<3>(lowerBound, upperBound));
  DomainT<3> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<3>(distFusedIndexSpace));
  RegionRequirement aReq = RegionRequirement(aPartition, 0, LEGION_REDOP_SUM_FLOAT64, LEGION_SIMULTANEOUS, get_logical_region(a));
  aReq.add_field(FID_VAL);
  RegionRequirement bReq = RegionRequirement(bPartition, 0, READ_ONLY, EXCLUSIVE, get_logical_region(b));
  bReq.add_field(FID_VAL);
  RegionRequirement cReq = RegionRequirement(cPartition, 0, READ_ONLY, EXCLUSIVE, get_logical_region(c));
  cReq.add_field(FID_VAL);
  task_4Args taskArgsRaw;
  taskArgsRaw.gridDim = gridDim;
  TaskArgument taskArgs = TaskArgument(&taskArgsRaw, sizeof(task_4Args));
  IndexLauncher launcher = IndexLauncher(taskID(4), domain, taskArgs, ArgumentMap());
  launcher.add_region_requirement(aReq);
  launcher.add_region_requirement(bReq);
  launcher.add_region_requirement(cReq);
  launcher.tag = launcher.tag | TACOMapper::UNTRACK_VALID_REGIONS;
  runtime->execute_index_space(ctx, launcher);

}
void registerTacoTasks() {
  {
    TaskVariantRegistrar registrar(taskID(1), "task_1");
    registrar.add_constraint(ProcessorConstraint(Processor::TOC_PROC));
    registrar.set_leaf();
    Runtime::preregister_task_variant<task_1>(registrar, "task_1");
  }
  {
    TaskVariantRegistrar registrar(taskID(2), "task_2");
    registrar.add_constraint(ProcessorConstraint(Processor::TOC_PROC));
    registrar.set_leaf();
    Runtime::preregister_task_variant<task_2>(registrar, "task_2");
  }
  {
    TaskVariantRegistrar registrar(taskID(3), "task_3");
    registrar.add_constraint(ProcessorConstraint(Processor::TOC_PROC));
    registrar.set_leaf();
    Runtime::preregister_task_variant<task_3>(registrar, "task_3");
  }
  {
    TaskVariantRegistrar registrar(taskID(4), "task_4");
    registrar.add_constraint(ProcessorConstraint(Processor::TOC_PROC));
    registrar.set_leaf();
    Runtime::preregister_task_variant<task_4>(registrar, "task_4");
  }
}
