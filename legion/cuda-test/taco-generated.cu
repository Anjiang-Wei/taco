#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "cudalibs.h"
#include "taco_legion_header.h"
#include "taco_mapper.h"
#define TACO_MIN(_a,_b) ((_a) < (_b) ? (_a) : (_b))
using namespace Legion;
typedef FieldAccessor<READ_ONLY,int32_t,1,coord_t,Realm::AffineAccessor<int32_t,1,coord_t>> AccessorROint32_t1;
typedef FieldAccessor<READ_WRITE,int32_t,1,coord_t,Realm::AffineAccessor<int32_t,1,coord_t>> AccessorRWint32_t1;

struct task_1Args {
  int32_t b1_dimension;
};

__global__
void task_1DeviceKernel0(AccessorRWint32_t1 a_vals, AccessorROint32_t1 b_vals, int32_t b1_dimension, int32_t in) {

  int32_t bvar = blockIdx.x;
  int32_t tvar = (threadIdx.x % (32));
  int32_t wvar = (threadIdx.x / 32);
  if (threadIdx.x >= 256) {
    return;
  }

  for (int32_t f3 = 0; f3 < 8; f3++) {
    int32_t f2 = tvar * 8 + f3;
    int32_t f1 = wvar * 256 + f2;
    int32_t il = bvar * 2048 + f1;
    int32_t i = in * ((b1_dimension + 3) / 4) + il;
    Point<1> a_access_point = Point<1>(i);
    Point<1> b_access_point = Point<1>(i);
    if (i >= b1_dimension)
      break;

    if (i >= (in + 1) * ((b1_dimension + 3) / 4))
      break;

    a_vals[a_access_point] = b_vals[b_access_point];
  }
}

void task_1(const Task* task, const std::vector<PhysicalRegion>& regions, Context ctx, Runtime* runtime) {
  PhysicalRegion a = regions[0];
  PhysicalRegion b = regions[1];

  int32_t in = task->index_point[0];
  task_1Args* args = (task_1Args*)(task->args);
  int32_t b1_dimension = args->b1_dimension;

  AccessorROint32_t1 b_vals(b, FID_VAL);
  AccessorRWint32_t1 a_vals(a, FID_VAL);

  task_1DeviceKernel0<<<(((b1_dimension + 3) / 4 + 2047) / 2048), (32 * 8)>>>(a_vals, b_vals, b1_dimension, in);
}

void computeLegion(Context ctx, Runtime* runtime, LogicalRegion a, LogicalRegion b) {
  auto a_index_space = get_index_space(a);
  int b1_dimension = runtime->get_index_space_domain(get_index_space(b)).hi()[0] + 1;
  auto b_index_space = get_index_space(b);

  Point<1> lowerBound = Point<1>(0);
  Point<1> upperBound = Point<1>(3);
  auto inIndexSpace = runtime->create_index_space(ctx, Rect<1>(lowerBound, upperBound));
  DomainT<1> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<1>(inIndexSpace));
  auto aDomain = runtime->get_index_space_domain(ctx, a_index_space);
  auto bDomain = runtime->get_index_space_domain(ctx, b_index_space);
  DomainPointColoring aColoring = DomainPointColoring();
  DomainPointColoring bColoring = DomainPointColoring();
  for (PointInDomainIterator<1> itr = PointInDomainIterator<1>(domain); itr.valid(); itr++) {
    int32_t in = (*itr)[0];
    Point<1> aStart = Point<1>((in * ((b1_dimension + 3) / 4)));
    Point<1> aEnd = Point<1>(TACO_MIN((in * ((b1_dimension + 3) / 4) + ((b1_dimension + 3) / 4 - 1)),aDomain.hi()[0]));
    Rect<1> aRect = Rect<1>(aStart, aEnd);
    if (!aDomain.contains(aRect.lo) || !aDomain.contains(aRect.hi)) aRect = aRect.make_empty();

    aColoring[(*itr)] = aRect;
    Point<1> bStart = Point<1>((in * ((b1_dimension + 3) / 4)));
    Point<1> bEnd = Point<1>(TACO_MIN((in * ((b1_dimension + 3) / 4) + ((b1_dimension + 3) / 4 - 1)),bDomain.hi()[0]));
    Rect<1> bRect = Rect<1>(bStart, bEnd);
    if (!bDomain.contains(bRect.lo) || !bDomain.contains(bRect.hi)) bRect = bRect.make_empty();

    bColoring[(*itr)] = bRect;
  }
  auto aPartition = runtime->create_index_partition(ctx, a_index_space, domain, aColoring, LEGION_DISJOINT_KIND);
  auto bPartition = runtime->create_index_partition(ctx, b_index_space, domain, bColoring, LEGION_DISJOINT_KIND);
  LogicalPartition aLogicalPartition = runtime->get_logical_partition(ctx, get_logical_region(a), aPartition);
  RegionRequirement aReq = RegionRequirement(aLogicalPartition, 0, READ_WRITE, EXCLUSIVE, get_logical_region(a));
  aReq.add_field(FID_VAL);
  LogicalPartition bLogicalPartition = runtime->get_logical_partition(ctx, get_logical_region(b), bPartition);
  RegionRequirement bReq = RegionRequirement(bLogicalPartition, 0, READ_ONLY, EXCLUSIVE, get_logical_region(b));
  bReq.add_field(FID_VAL);
  task_1Args taskArgsRaw;
  taskArgsRaw.b1_dimension = b1_dimension;
  TaskArgument taskArgs = TaskArgument(&taskArgsRaw, sizeof(task_1Args));
  IndexLauncher launcher = IndexLauncher(taskID(1), domain, taskArgs, ArgumentMap());
  launcher.add_region_requirement(aReq);
  launcher.add_region_requirement(bReq);
  auto fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();

}
void registerTacoTasks() {
  {
    TaskVariantRegistrar registrar(taskID(1), "task_1");
    registrar.add_constraint(ProcessorConstraint(Processor::LOC_PROC));
    registrar.set_leaf();
    Runtime::preregister_task_variant<task_1>(registrar, "task_1");
  }
}
