#include "hip/hip_runtime.h"
#include "taco_legion_header.h"
#include "taco_mapper.h"
#define TACO_MIN(_a,_b) ((_a) < (_b) ? (_a) : (_b))
#define TACO_MAX(_a,_b) ((_a) < (_b) ? (_b) : (_a))
using namespace Legion;

#include "taco-generated.cuh"
#include "hipblas.h"
#include "hipsparse.h"
#include "cudalibs.h"
#include "leaf_kernels.cuh"
typedef FieldAccessor<READ_ONLY,double,1,coord_t,Realm::AffineAccessor<double,1,coord_t>> AccessorROdouble1;
typedef FieldAccessor<READ_ONLY,double,2,coord_t,Realm::AffineAccessor<double,2,coord_t>> AccessorROdouble2;
typedef FieldAccessor<READ_WRITE,double,1,coord_t,Realm::AffineAccessor<double,1,coord_t>> AccessorRWdouble1;
typedef FieldAccessor<READ_ONLY,int32_t,1,coord_t,Realm::AffineAccessor<int32_t,1,coord_t>> AccessorROint32_t1;
typedef FieldAccessor<READ_ONLY,Rect<1>,1,coord_t,Realm::AffineAccessor<Rect<1>,1,coord_t>> AccessorRORect_1_1;

struct task_1Args {
  Legion::FieldID A_vals_field_id;
  int64_t B2Size;
  Legion::FieldID B2_indices_field_id_1_0;
  Legion::FieldID B2_indices_field_id_1_1;
  Legion::FieldID B_vals_field_id;
  int64_t C2_dimension;
  Legion::FieldID C_vals_field_id;
  Legion::FieldID D_vals_field_id;
  int32_t pieces;
};


partitionPackForcomputeLegion partitionForcomputeLegion(Legion::Context ctx, Legion::Runtime* runtime, LegionTensor* A, LegionTensor* B, LegionTensor* C, LegionTensor* D, int32_t pieces) {
  auto A2_pos_parent = A->indicesParents[1][0];
  auto A2_crd_parent = A->indicesParents[1][1];
  RegionWrapper A_vals = A->vals;
  IndexSpace A_dense_run_0 = A->denseLevelRuns[0];
  RegionWrapper B2_pos = B->indices[1][0];
  RegionWrapper B2_crd = B->indices[1][1];
  auto B2_pos_parent = B->indicesParents[1][0];
  RegionWrapper B_vals = B->vals;
  IndexSpace B_dense_run_0 = B->denseLevelRuns[0];
  auto B2_indices_field_id_1_0 = B->indicesFieldIDs[1][0];
  RegionWrapper C_vals = C->vals;
  IndexSpace C_dense_run_0 = C->denseLevelRuns[0];

  auto computePartitions = partitionPackForcomputeLegion();

  int64_t B2Size = runtime->get_index_space_domain(ctx, get_index_space(B2_crd)).hi()[0] + 1;

  Point<1> lowerBound = Point<1>(0);
  Point<1> upperBound = Point<1>((pieces - 1));
  auto fposoIndexSpace = runtime->create_index_space(ctx, Rect<1>(lowerBound, upperBound));
  DomainT<1> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<1>(fposoIndexSpace));
  DomainT<1> B2_crd_domain = runtime->get_index_space_domain(ctx, B2_crd.get_index_space());
  DomainPointColoring B2_crd_coloring = DomainPointColoring();
  for (PointInDomainIterator<1> itr = PointInDomainIterator<1>(domain); itr.valid(); itr++) {
    int64_t fposo = (*itr)[0];
    Point<1> B2CrdStart = Point<1>((fposo * ((B2Size + (pieces - 1)) / pieces)));
    Point<1> B2CrdEnd = Point<1>(TACO_MIN((fposo * ((B2Size + (pieces - 1)) / pieces) + ((B2Size + (pieces - 1)) / pieces - 1)),B2_crd_domain.bounds.hi[0]));
    Rect<1> B2CrdRect = Rect<1>(B2CrdStart, B2CrdEnd);
    if (!B2_crd_domain.contains(B2CrdRect.lo) || !B2_crd_domain.contains(B2CrdRect.hi)) {
      B2CrdRect = B2CrdRect.make_empty();
    }
    B2_crd_coloring[(*itr)] = B2CrdRect;
  }
  IndexPartition B2_crd_index_part = runtime->create_index_partition(ctx, B2_crd.get_index_space(), domain, B2_crd_coloring, LEGION_COMPUTE_KIND);
  Legion::LogicalPartition B2_crd_part = runtime->get_logical_partition(ctx, B2_crd, B2_crd_index_part);
  IndexPartition posSparsePartB2 = runtime->create_partition_by_preimage_range(
    ctx,
    B2_crd_index_part,
    B2_pos,
    B2_pos_parent,
    B2_indices_field_id_1_0,
    runtime->get_index_partition_color_space_name(ctx, B2_crd_index_part),
    LEGION_ALIASED_INCOMPLETE_KIND
  );
  IndexPartition posIndexPartB2 = densifyPartition(ctx, runtime, get_index_space(B2_pos), posSparsePartB2);
  Legion::LogicalPartition posPartB2 = runtime->get_logical_partition(ctx, B2_pos, posIndexPartB2);
  Legion::LogicalPartition BValsLogicalPart = copyPartition(ctx, runtime, B2_crd_part, B_vals);
  IndexPartition BDenseRun0Partition = copyPartition(ctx, runtime, posPartB2, B_dense_run_0);
  Legion::LogicalPartition AValsLogicalPart = copyPartition(ctx, runtime, B2_crd_part, A_vals);
  Legion::LogicalPartition posPartA2 = copyPartition(ctx, runtime, posPartB2, A2_pos_parent);
  Legion::LogicalPartition crdPartA2 = copyPartition(ctx, runtime, B2_crd_part, A2_crd_parent);
  IndexPartition ADenseRun0Partition = copyPartition(ctx, runtime, posPartA2, A_dense_run_0);
  IndexPartition CDenseRun0Partition = AffineProjection(0).apply(ctx, runtime, BDenseRun0Partition, C_dense_run_0);
  auto C_vals_partition = copyPartition(ctx, runtime, CDenseRun0Partition, get_logical_region(C_vals));
  computePartitions.APartition.indicesPartitions = std::vector<std::vector<Legion::LogicalPartition>>(2);
  computePartitions.APartition.denseLevelRunPartitions = std::vector<IndexPartition>(2);
  computePartitions.APartition.indicesPartitions[1].push_back(posPartA2);
  computePartitions.APartition.indicesPartitions[1].push_back(crdPartA2);
  computePartitions.APartition.valsPartition = AValsLogicalPart;
  computePartitions.APartition.denseLevelRunPartitions[0] = ADenseRun0Partition;
  computePartitions.BPartition.indicesPartitions = std::vector<std::vector<Legion::LogicalPartition>>(2);
  computePartitions.BPartition.denseLevelRunPartitions = std::vector<IndexPartition>(2);
  computePartitions.BPartition.indicesPartitions[1].push_back(posPartB2);
  computePartitions.BPartition.indicesPartitions[1].push_back(B2_crd_part);
  computePartitions.BPartition.valsPartition = BValsLogicalPart;
  computePartitions.BPartition.denseLevelRunPartitions[0] = BDenseRun0Partition;
  computePartitions.CPartition.indicesPartitions = std::vector<std::vector<Legion::LogicalPartition>>(2);
  computePartitions.CPartition.denseLevelRunPartitions = std::vector<IndexPartition>(2);
  computePartitions.CPartition.valsPartition = C_vals_partition;
  computePartitions.CPartition.denseLevelRunPartitions[0] = CDenseRun0Partition;

  return computePartitions;
}

__global__
void task_1DeviceKernel0(int64_t B2Size, int64_t fposo, int64_t* i_blockStarts, int32_t pieces, int64_t pointID1, AccessorRORect_1_1 B2_pos_accessor, AccessorROint32_t1 B2_crd_accessor, AccessorRWdouble1 A_vals_rw_accessor, AccessorROdouble1 B_vals_ro_accessor, AccessorROdouble2 C_vals_ro_accessor, AccessorROdouble2 D_vals_ro_accessor, Legion::FieldID A_vals_field_id, Legion::FieldID B2_indices_field_id_1_0, Legion::FieldID B2_indices_field_id_1_1, Legion::FieldID B_vals_field_id, int64_t C2_dimension, Legion::FieldID C_vals_field_id, Legion::FieldID D_vals_field_id) {

  int64_t block = blockIdx.x;
  int64_t thread = (threadIdx.x % (32));
  int64_t warp = (threadIdx.x / 32);
  if (threadIdx.x >= 256) {
    return;
  }

  int64_t pointID2 = pointID1 * (((B2Size + (pieces - 1)) / pieces + 2047) / 2048) + block;
  int64_t pointID3 = pointID2 * 8 + warp;
  int64_t pB2_begin = i_blockStarts[block];
  int64_t pB2_end = i_blockStarts[(block + 1)];
  int64_t fposi1 = warp * 256;
  int64_t fposi = block * 2048 + fposi1;
  int64_t fposB = fposo * ((B2Size + (pieces - 1)) / pieces) + fposi;
  int64_t i_pos = taco_binarySearchBefore(B2_pos_accessor, pB2_begin, pB2_end, fposB);
  int64_t i = i_pos;
  for (int64_t nnz = 0; nnz < 256; nnz++) {
    int64_t fposi1 = warp * 256 + nnz;
    int64_t fposi = block * 2048 + fposi1;
    int64_t fposB = fposo * ((B2Size + (pieces - 1)) / pieces) + fposi;
    if (fposB >= (fposo + 1) * ((B2Size + (pieces - 1)) / pieces))
      break;

    if (fposB >= B2Size)
      break;

    int64_t f = B2_crd_accessor[fposB];
    while (!(B2_pos_accessor[i_pos].contains(fposB))) {
      i_pos = i_pos + 1;
      i = i_pos;
    }
    int64_t pointID4 = pointID3 * 32 + thread;
    for (int64_t dense_b = 0; dense_b < ((C2_dimension + 31) / 32); dense_b++) {
      int64_t j = dense_b * 32 + thread;
      if (j >= C2_dimension)
        break;

      atomicAddWarp(A_vals_rw_accessor.ptr(Point<1>(fposB)), flattenPoint(A_vals_rw_accessor, Point<1>(fposB)), ((B_vals_ro_accessor[Point<1>(fposB)] * C_vals_ro_accessor[Point<2>(i, j)]) * D_vals_ro_accessor[Point<2>(j, f)]));
    }
  }
}

void task_1(const Task* task, const std::vector<PhysicalRegion>& regions, Context ctx, Runtime* runtime) {
  PhysicalRegion A2_pos = regions[0];
  LogicalRegion A2_pos_parent = regions[0].get_logical_region();
  PhysicalRegion A2_crd = regions[1];
  LogicalRegion A2_crd_parent = regions[1].get_logical_region();
  PhysicalRegion A_vals = regions[2];
  LogicalRegion A_vals_parent = regions[2].get_logical_region();
  PhysicalRegion B2_pos = regions[3];
  LogicalRegion B2_pos_parent = regions[3].get_logical_region();
  PhysicalRegion B2_crd = regions[4];
  LogicalRegion B2_crd_parent = regions[4].get_logical_region();
  PhysicalRegion B_vals = regions[5];
  LogicalRegion B_vals_parent = regions[5].get_logical_region();
  PhysicalRegion C_vals = regions[6];
  LogicalRegion C_vals_parent = regions[6].get_logical_region();
  PhysicalRegion D_vals = regions[7];
  LogicalRegion D_vals_parent = regions[7].get_logical_region();

  int64_t fposo = task->index_point[0];
  task_1Args* args = (task_1Args*)(task->args);
  Legion::FieldID A_vals_field_id = args->A_vals_field_id;
  int64_t B2Size = args->B2Size;
  Legion::FieldID B2_indices_field_id_1_0 = args->B2_indices_field_id_1_0;
  Legion::FieldID B2_indices_field_id_1_1 = args->B2_indices_field_id_1_1;
  Legion::FieldID B_vals_field_id = args->B_vals_field_id;
  int64_t C2_dimension = args->C2_dimension;
  Legion::FieldID C_vals_field_id = args->C_vals_field_id;
  Legion::FieldID D_vals_field_id = args->D_vals_field_id;
  int32_t pieces = args->pieces;

  auto B_vals_ro_accessor = createAccessor<AccessorROdouble1>(B_vals, B_vals_field_id);
  auto C_vals_ro_accessor = createAccessor<AccessorROdouble2>(C_vals, C_vals_field_id);
  auto D_vals_ro_accessor = createAccessor<AccessorROdouble2>(D_vals, D_vals_field_id);
  auto A_vals_rw_accessor = createAccessor<AccessorRWdouble1>(A_vals, A_vals_field_id);
  auto B2_pos_accessor = createAccessor<AccessorRORect_1_1>(B2_pos, B2_indices_field_id_1_0);
  auto B2_crd_accessor = createAccessor<AccessorROint32_t1>(B2_crd, B2_indices_field_id_1_1);

  if (runtime->get_index_space_domain(ctx, get_index_space(B2_crd)).empty())
    return ;

  DomainT<1> B2PosDomain = runtime->get_index_space_domain(ctx, get_index_space(B2_pos));
  DomainT<1> B2CrdDomain = runtime->get_index_space_domain(ctx, get_index_space(B2_crd));
  Legion::DeferredBuffer<int64_t, 1> buf = Legion::DeferredBuffer<int64_t, 1>(Rect<1>(0, (((B2Size + (pieces - 1)) / pieces + 2047) / 2048)), Legion::Memory::Kind::GPU_FB_MEM);
  int64_t* i_blockStarts = buf.ptr(0);
  taco_binarySearchBeforeBlockLaunch(
    B2_pos_accessor,
    i_blockStarts,
    B2PosDomain.bounds.lo,
    B2PosDomain.bounds.hi,
    2048,
    256,
    (((B2Size + (pieces - 1)) / pieces + 2047) / 2048),
    B2CrdDomain.bounds.lo
  );
  int64_t pointID1 = fposo + TACO_PARTITION_COLOR_OFFSET;
  if ((((B2Size + (pieces - 1)) / pieces + 2047) / 2048) > 0) {
    task_1DeviceKernel0<<<(((B2Size + (pieces - 1)) / pieces + 2047) / 2048), (32 * 8)>>>(B2Size, fposo, i_blockStarts, pieces, pointID1, B2_pos_accessor, B2_crd_accessor, A_vals_rw_accessor, B_vals_ro_accessor, C_vals_ro_accessor, D_vals_ro_accessor, A_vals_field_id, B2_indices_field_id_1_0, B2_indices_field_id_1_1, B_vals_field_id, C2_dimension, C_vals_field_id, D_vals_field_id);
  }
}

void computeLegion(Legion::Context ctx, Legion::Runtime* runtime, LegionTensor* A, LegionTensor* B, LegionTensor* C, LegionTensor* D, partitionPackForcomputeLegion* partitionPack, int32_t pieces) {
  auto A2_pos_parent = A->indicesParents[1][0];
  auto A2_crd_parent = A->indicesParents[1][1];
  auto A_vals_parent = A->valsParent;
  auto A_vals_field_id = A->valsFieldID;
  auto A2_indices_field_id_1_0 = A->indicesFieldIDs[1][0];
  auto A2_indices_field_id_1_1 = A->indicesFieldIDs[1][1];
  RegionWrapper B2_crd = B->indices[1][1];
  auto B2_pos_parent = B->indicesParents[1][0];
  auto B2_crd_parent = B->indicesParents[1][1];
  auto B_vals_parent = B->valsParent;
  auto B_vals_field_id = B->valsFieldID;
  auto B2_indices_field_id_1_0 = B->indicesFieldIDs[1][0];
  auto B2_indices_field_id_1_1 = B->indicesFieldIDs[1][1];
  size_t C2_dimension = C->dims[1];
  auto C_vals_parent = C->valsParent;
  auto C_vals_field_id = C->valsFieldID;
  RegionWrapper D_vals = D->vals;
  auto D_vals_parent = D->valsParent;
  auto D_vals_field_id = D->valsFieldID;

  int64_t B2Size = runtime->get_index_space_domain(ctx, get_index_space(B2_crd)).hi()[0] + 1;

  Point<1> lowerBound = Point<1>(0);
  Point<1> upperBound = Point<1>((pieces - 1));
  auto fposoIndexSpace = runtime->create_index_space(ctx, Rect<1>(lowerBound, upperBound));
  DomainT<1> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<1>(fposoIndexSpace));
  task_1Args taskArgsRaw1;
  taskArgsRaw1.A_vals_field_id = A_vals_field_id;
  taskArgsRaw1.B2Size = B2Size;
  taskArgsRaw1.B2_indices_field_id_1_0 = B2_indices_field_id_1_0;
  taskArgsRaw1.B2_indices_field_id_1_1 = B2_indices_field_id_1_1;
  taskArgsRaw1.B_vals_field_id = B_vals_field_id;
  taskArgsRaw1.C2_dimension = C2_dimension;
  taskArgsRaw1.C_vals_field_id = C_vals_field_id;
  taskArgsRaw1.D_vals_field_id = D_vals_field_id;
  taskArgsRaw1.pieces = pieces;
  TaskArgument taskArgs = TaskArgument(&taskArgsRaw1, sizeof(task_1Args));
  IndexLauncher launcher = IndexLauncher(taskID(1), domain, taskArgs, ArgumentMap());
  launcher.add_region_requirement(RegionRequirement(
    partitionPack->APartition.indicesPartitions[1][0],
    0,
    READ_ONLY,
    EXCLUSIVE,
    get_logical_region(A2_pos_parent),
    Mapping::DefaultMapper::VIRTUAL_MAP
  ).add_field(A2_indices_field_id_1_0));
  launcher.add_region_requirement(RegionRequirement(
    partitionPack->APartition.indicesPartitions[1][1],
    0,
    READ_ONLY,
    EXCLUSIVE,
    get_logical_region(A2_crd_parent),
    Mapping::DefaultMapper::VIRTUAL_MAP
  ).add_field(A2_indices_field_id_1_1));
  launcher.add_region_requirement(RegionRequirement(partitionPack->APartition.valsPartition, 0, READ_WRITE, EXCLUSIVE, A_vals_parent).add_field(A_vals_field_id));
  launcher.add_region_requirement(RegionRequirement(partitionPack->BPartition.indicesPartitions[1][0], 0, READ_ONLY, EXCLUSIVE, get_logical_region(B2_pos_parent)).add_field(B2_indices_field_id_1_0));
  launcher.add_region_requirement(RegionRequirement(partitionPack->BPartition.indicesPartitions[1][1], 0, READ_ONLY, EXCLUSIVE, get_logical_region(B2_crd_parent)).add_field(B2_indices_field_id_1_1));
  launcher.add_region_requirement(RegionRequirement(partitionPack->BPartition.valsPartition, 0, READ_ONLY, EXCLUSIVE, B_vals_parent).add_field(B_vals_field_id));
  launcher.add_region_requirement(RegionRequirement(partitionPack->CPartition.valsPartition, 0, READ_ONLY, EXCLUSIVE, C_vals_parent).add_field(C_vals_field_id));
  launcher.add_region_requirement(RegionRequirement(get_logical_region(D_vals), READ_ONLY, EXCLUSIVE, D_vals_parent).add_field(D_vals_field_id));
  launcher.tag = launcher.tag | TACOMapper::UNTRACK_VALID_REGIONS;
  runtime->execute_index_space(ctx, launcher);

}
void registerTacoTasks() {
  {
    TaskVariantRegistrar registrar(taskID(1), "task_1");
    registrar.add_constraint(ProcessorConstraint(Processor::TOC_PROC));
    registrar.set_leaf();
    Runtime::preregister_task_variant<task_1>(registrar, "task_1");
  }
}
